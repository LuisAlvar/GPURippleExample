#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "cpu_anim.h"

#define DIM 1280

/// <summary>
/// 
/// </summary>
/// <param name="ptr">- pointer to device memory that holds the output pixels</param>
/// <param name="ticks">- the current animation time so it can generate the correct frame</param>
/// <returns></returns>
__global__ void kernel(unsigned char* ptr, int ticks)
{
	//Finding the x,y coordinates
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	//Using (x,y) to determine the linearize index 
	int linear_offset = x + y * blockDim.x * gridDim.x;

	//now caluclate the value at that position 
	float fx = x - DIM / 2;
	float fy = y - DIM / 2;
	float d = sqrtf(fx * fx + fy * fy);

	unsigned char grey = (unsigned char)(128.0f + 127.0f * cos(d / 10.0f - ticks / 7.0f) / (d / 10.0f + 1.0f));

	ptr[linear_offset * 4 + 0] = grey;
	ptr[linear_offset * 4 + 1] = grey;
	ptr[linear_offset * 4 + 2] = grey;
	ptr[linear_offset * 4 + 3] = 255;
	 
}


struct DataBlock {
	unsigned char* dev_bitmap;
	CPUAnimBitmap* bitmap;
};

// clean up memory allocated on the GPU
void cleanup(DataBlock *d){
	hipFree(d->dev_bitmap);
}

//This function will be called by the strucutre every time it wants to generate a new frame of the animation.  
void generate_frame(DataBlock *d, int ticks) {
	dim3 blocks(DIM/16, DIM/16);
	dim3 threads(16, 16);

	kernel<<< blocks, threads >>>(d->dev_bitmap, ticks);

	hipMemcpy(d->bitmap->get_ptr(), d->dev_bitmap, d->bitmap->image_size(), hipMemcpyDeviceToHost);
}

int main( void )
{

	DataBlock data;
	CPUAnimBitmap bitmap(DIM,DIM, &data);
	data.bitmap = &bitmap;

	hipMalloc( (void**)&data.dev_bitmap, bitmap.image_size() );

	// We pass a function pointer to generate_frame() 
	bitmap.anim_and_exit((void (*)(void*, int))generate_frame, (void (*)(void*))cleanup);

	return 0;
}
